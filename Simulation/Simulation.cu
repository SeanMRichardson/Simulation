#include "hip/hip_runtime.h"
#ifndef _SIMPLETEXTURE3D_KERNEL_CU_
#define _SIMPLETEXTURE3D_KERNEL_CU_

#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>

#include <GL/glew.h>
#include <glm/glm.hpp>
#include <glm/gtc/matrix_transform.hpp>
#include <glm/gtc/type_ptr.hpp>

#include <hip/hip_runtime_api.h>         // helper functions for CUDA error check
#include <helper_gl.h>
#include <hip/hip_vector_types.h>

//Round a / b to nearest higher integer value
int cuda_iDivUp(int a, int b)
{
	return (a + (b - 1)) / b;
}

__global__ void
calculate_vertices_kernel(glm::vec3* vertices, GLbyte* data, int width, int height)
{
	//GLuint x = (blockIdx.x * blockDim.x) + threadIdx.x;
	//GLuint z = (blockIdx.y * blockDim.y) + threadIdx.y;

	GLuint x = blockIdx.x;
	GLuint z = blockIdx.y;

	int offset = (z * height) + x;

	if (x < width && z < height)
	{
		vertices[offset].x = x;
		vertices[offset].y = abs(data[offset]*0.4)-3.6f; //reading in the heights from the data, and applying offsets to make it more readable
		vertices[offset].z = z;
	}
}

__global__ void
calculate_indices_kernel(GLuint* indices, GLint numIndices, int width, int height)
{
	GLuint x = blockIdx.x;
	GLuint y = blockIdx.y;

	int offset = (y * height * 6) + (x * 6);

	if (x < width && y < height)
	{
		long a = (x * (width)) + y;
		long b = ((x + 1) * (width)) + y;
		long c = ((x + 1) * (width)) + (y + 1);
		long d = (x * (width)) + (y + 1);

		indices[offset] = c;
		indices[offset + 1] = b;
		indices[offset + 2] = a;

		indices[offset + 3] = a;
		indices[offset + 4] = d;
		indices[offset + 5] = c;
	}
}

__global__ void
calculate_normals_kernel(glm::vec3* normals, GLuint* indices, glm::vec3* vertices, int width, int height)
{
	GLuint x = blockIdx.x;
	GLuint y = blockIdx.y;

	int offset = (y * height * 3) + (x* 3);

	unsigned int a = indices[offset];
	unsigned int b = indices[offset + 1];
	unsigned int c = indices[offset + 2];

	glm::vec3 normal = cross((vertices[b] - vertices[c]), (vertices[a] - vertices[c]));

	normals[a] += normal;
	normals[b] += normal;
	normals[c] += normal;
}

/*
Process each vertex from the raw file in its own block
Each block needs just one thread
*/
extern "C" void CalculateVertices(glm::vec3* vertices, GLbyte* data, int width, int height)
{
	dim3 grid(width, height);
	dim3 block(1, 1, 1);
	calculate_vertices_kernel << <grid, block >> > (vertices, data, width, height);
}

/*
Each block in the grid represents the space between 4 vertices
Each space is split into 2 triangles
*/
extern "C" void CalculateIndices(GLuint* indices, GLint numIndices, int width, int height)
{

	dim3 block(1, 1, 1);
	dim3 grid(width-1, width-1);
	calculate_indices_kernel << <grid, block >> > (indices, numIndices, width, height);
}

/*
The surface normal of each triangle is processed in its own block/thread
There are 2 triangles per set of 4 vertices so we create a grid with twice the dimensions
*/
extern "C" void CalculateNormals(glm::vec3* normals, GLuint* indices, glm::vec3* vertices, int width, int height)
{
	dim3 block(1, 1, 1);
	dim3 grid(width*2, height*2);
	calculate_normals_kernel << <grid, block >> > (normals, indices, vertices, width, height);
}
#endif